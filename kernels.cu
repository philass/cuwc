#include <hip/hip_runtime.h>


/*
   Unoptimized LineCount kernel implementation
   Should be able to get 20x speed up
*/
__global__ void lineCount(char *g_idata, int *g_odata) {
  extern __shared__ int sdata[];
  // each thread loads one element from global to shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
  sdata[tid] = g_idata[i] == '\n' ? 1 : 0; 
  __syncthreads();
  // do reduction in shared mem
  for (unsigned int s=1; s < blockDim.x; s *= 2) {
    if (tid % (2*s) == 0) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }
  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

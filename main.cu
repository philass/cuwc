#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include "kernels.cuh"



int main(int argc, char *argv[]) {
  int fileStarts;
  if (argc == 1) {
    std::cout << "Missing arguments" << std::endl;
    return 0;
  }
  bool get_c; bool get_w; bool get_l;
  if (argv[1][0] != '-') {
    get_c = true;
    get_w = true;
    get_l = true;
    fileStarts = 1;
  } else {
    get_c = false;
    get_w = false;
    get_l = false;
    for (int i = 1; i < argc; i++) {
      if (argv[i][0] == '-') {
        int j = 1;
        while (argv[i][j] != '\0') {
          switch (argv[i][j]) {
            case 'c':
              get_c = true;
              break;
            case 'w':
              get_w = true;
              break;
            case 'l':
              get_l = true;
              break;
            default:
              std::cout << "Unrecognized option : -" << argv[i][j] << std::endl;
              return 1;
          }
          j++;
        }
      } else {
        fileStarts = i;
        break;
      }
    }
  }

  // READ FILE
  FILE *fp;
  std::string firstFile = argv[fileStarts];
  fp = fopen (argv[fileStarts], "rb");
  char* string = NULL;
  size_t len;
  ssize_t file_length = getdelim( &string, &len, '\0', fp);
  // Check if file reading failed
  if (file_length == -1) {
    std::cout << "Couldn't read file!" << std::endl;
    return 1;
  }

  // Make GPU allocations
  size_t mem_size = sizeof(char) * file_length;
  char* d_in;
  int* d_out;
  hipMalloc((void**)&d_in, mem_size);
  hipMalloc((void**)&d_out, mem_size);

  // Copy to GPU Memory
  hipMemcpy(d_in, string, mem_size, hipMemcpyHostToDevice);

  // Call Kernel
  int numBlocks = file_length / 1024 + 1;
  reduce0<<<numBlocks, 1024 >>>(d_in, d_out);
  hipDeviceSynchronize();
  
  // Get the result in host memory
  int* h_out = (int*) malloc(sizeof(int) * 10);
  hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
  
  // Sum the results from different blocks
  int sum;
  for (int i = 0; i < numBlocks; i++) {
    sum += h_out[i];
  }

  // Print the results
  if (get_l) std::cout << sum << " " << firstFile << std::endl;
}


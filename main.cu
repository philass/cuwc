#include <iostream>
#include <fstream>
#include <unistd.h>
#include <string>
#include <hip/hip_runtime.h>
#include "kernels.cuh"

int getSum(int* values, int size) {
  int sum = 0;
  for (int i = 0; i < size; i++) {
    sum += values[i];
  }
  return sum;
}

int main(int argc, char *argv[]) {
  if (argc == 1) {
    std::cout << "Missing arguments" << std::endl;
    return 0;
  }
  int opt;
  bool get_c = false, get_w = false, get_l = false;
  while ((opt = getopt(argc, argv, "cwl")) != -1) {
    switch (opt) {
      case 'c': get_c = true; break;
      case 'w': get_w = true; break;
      case 'l': get_l = true; break;
      default: 
        fprintf(stderr, "Usage: %s [-cwl] [file...]\n", argv[0]);
        exit(EXIT_FAILURE);
    }
  }
  if (optind == 1) get_c = get_w = get_l = true;

  // READ FILE
  FILE *fp;
  fp = fopen (argv[optind], "rb");
  char* string = NULL;
  size_t len;
  int file_length = getdelim( &string, &len, '\0', fp);
  // Check if file reading failed
  if (file_length == -1) {
    std::cout << "Couldn't read file!" << std::endl;
    return 1;
  }

  int lineSum = 0;
  int wordSum = 0;
  if (get_l || get_w) {
    size_t mem_size = sizeof(char) * file_length;
    char* d_in;
    hipMalloc((void**)&d_in, mem_size);
    hipMemcpy(d_in, string, mem_size, hipMemcpyHostToDevice);

    int numBlocks = file_length / 1024 + 1;
    if (get_l) {
      int* d_out_lines;
      hipMalloc((void**)&d_out_lines, file_length * sizeof(int));
      reduceLines<<<numBlocks, 1024 >>>(d_in, d_out_lines, file_length);
      hipDeviceSynchronize();
      int* h_out_lines = (int*) malloc(sizeof(int) * numBlocks);
      hipMemcpy(h_out_lines, d_out_lines, mem_size, hipMemcpyDeviceToHost);
      lineSum = getSum(h_out_lines, numBlocks);
    }
    
    if (get_w) {
      int* d_out_words;
      hipMalloc((void**)&d_out_words, file_length * sizeof(int));
      reduceWords<<<numBlocks, 1024 >>>(d_in, d_out_words, file_length);
      hipDeviceSynchronize();
      int* h_out_words = (int*) malloc(sizeof(int) * numBlocks);
      hipMemcpy(h_out_words, d_out_words, mem_size, hipMemcpyDeviceToHost);
      wordSum = getSum(h_out_words, numBlocks);
    }
  }
  if (get_l) std::cout << lineSum << " ";
  if (get_w) std::cout << wordSum << " ";
  if (get_c) std::cout << file_length << " ";
  std::cout << argv[optind] << std::endl;
  return 0;
}


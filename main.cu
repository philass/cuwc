#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include "kernels.cu.h"

int main(int argc, char *argv[]) {
  int fileStarts;
  bool get_c; bool get_w; bool get_l;
  if (argv[1][0] != '-') {
    get_c = true;
    get_w = true;
    get_l = true;
    fileStarts = 1;
  } else {
    get_c = false;
    get_w = false;
    get_l = false;
    for (int i = 1; i < argc; i++) {
      if (argv[i][0] == '-') {
        int j = 1;
        while (argv[i][j] != '\0') {
          switch (argv[i][j]) {
            case 'c':
              get_c = true;
              break;
            case 'w':
              get_w = true;
              break;
            case 'l':
              get_l = true;
              break;
            default:
              std::cout << "Unrecognized option : -" << argv[i][j] << std::endl;
              return 1;
          }
          j++;
        }
      } else {
        fileStarts = i;
        break;
      }
    }
  }
  std::cout << "c, w, l -> " << get_c << " " << get_w << " " << get_l << std::endl;
  std::cout << "fileStarts -> " << fileStarts << " " << argv[fileStarts] << std::endl;

  std::string firstFile = argv[fileStarts];
  std::ifstream in(firstFile);
  std::string contents((std::istreambuf_iterator<char>(in)), 
  std::istreambuf_iterator<char>());
  int file_length = contents.length();
  const char* string = contents.c_str();

  //char string[] = "this \n is \n a \n \n test \ntew";
  //int vals[10] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  size_t mem_size = sizeof(char) * file_length;
  char* d_in;
  int* d_out;
  hipMalloc((void**)&d_in, mem_size);
  hipMalloc((void**)&d_out, mem_size);
  hipMemcpy(d_in, string, mem_size, hipMemcpyHostToDevice);
  reduce0<<<1, 512 >>>(d_in, d_out);
  hipDeviceSynchronize();
  int* h_out = (int*) malloc(sizeof(int) * 10);
  hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
  std::cout << h_out[0] << " " << firstFile << std::endl;
  std::cout << h_out[1] << std::endl;
  std::cout << h_out[2] << std::endl;
  std::cout << h_out[3] << std::endl;
}


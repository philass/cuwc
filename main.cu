#include <iostream>
#include <fstream>
#include <string>
#include <hip/hip_runtime.h>
#include "kernels.cuh"


int getSum(int* values, int size) {
  int sum = 0;
  for (int i = 0; i < size; i++) {
    sum += values[i];
  }
  return sum;
}


int main(int argc, char *argv[]) {
  int fileStarts;
  if (argc == 1) {
    std::cout << "Missing arguments" << std::endl;
    return 0;
  }
  bool get_c; bool get_w; bool get_l;
  if (argv[1][0] != '-') {
    get_c = true;
    get_w = true;
    get_l = true;
    fileStarts = 1;
  } else {
    get_c = false;
    get_w = false;
    get_l = false;
    for (int i = 1; i < argc; i++) {
      if (argv[i][0] == '-') {
        int j = 1;
        while (argv[i][j] != '\0') {
          switch (argv[i][j]) {
            case 'c':
              get_c = true;
              break;
            case 'w':
              get_w = true;
              break;
            case 'l':
              get_l = true;
              break;
            default:
              std::cout << "Unrecognized option : -" << argv[i][j] << std::endl;
              return 1;
          }
          j++;
        }
      } else {
        fileStarts = i;
        break;
      }
    }
  }

  // READ FILE
  FILE *fp;
  std::string firstFile = argv[fileStarts];
  fp = fopen (argv[fileStarts], "rb");
  char* string = NULL;
  size_t len;
  int file_length = getdelim( &string, &len, '\0', fp);
  // Check if file reading failed
  if (file_length == -1) {
    std::cout << "Couldn't read file!" << std::endl;
    return 1;
  }

  int lineSum = 0;
  int wordSum = 0;
  if (get_l || get_w) {
    size_t mem_size = sizeof(char) * file_length;
    char* d_in;
    hipMalloc((void**)&d_in, mem_size);
    hipMemcpy(d_in, string, mem_size, hipMemcpyHostToDevice);

    int numBlocks = file_length / 1024 + 1;
    if (get_l) {
      int* d_out_lines;
      hipMalloc((void**)&d_out_lines, file_length * sizeof(int));
      reduceLines<<<numBlocks, 1024 >>>(d_in, d_out_lines, file_length);
      hipDeviceSynchronize();
      int* h_out_lines = (int*) malloc(sizeof(int) * numBlocks);
      hipMemcpy(h_out_lines, d_out_lines, mem_size, hipMemcpyDeviceToHost);
      lineSum = getSum(h_out_lines, numBlocks);
    }
    
    if (get_w) {
      int* d_out_words;
      hipMalloc((void**)&d_out_words, file_length * sizeof(int));
      reduceWords<<<numBlocks, 1024 >>>(d_in, d_out_words, file_length);
      hipDeviceSynchronize();
      int* h_out_words = (int*) malloc(sizeof(int) * numBlocks);
      hipMemcpy(h_out_words, d_out_words, mem_size, hipMemcpyDeviceToHost);
      wordSum = getSum(h_out_words, numBlocks);
    }
  }
  
  if (get_l) std::cout << lineSum << " ";
  if (get_w) std::cout << wordSum << " ";
  if (get_c) std::cout << file_length << " ";
  std::cout << firstFile << std::endl;
 
}


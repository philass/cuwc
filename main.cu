#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "kernels.cu.h"

/*__global__ void reduce(int *input, int *output, unsigned int n)
{
    // Determine this thread's various ids
    unsigned int block_size = blockDim.x;
    unsigned int thread_id = threadIdx.x;
    unsigned int block_id = blockIdx.x;

    // Determine the number of values the threads in this block will need to operate upon
    // (remember, the last block may need fewer than the others).
    unsigned int chunk_size = (block_id * block_size * 2 + block_size * 2 > n) ? n % (block_size * 2) : block_size * 2;
    // How read the line above: if we're the last block and n is not divisible by
    // (block_size * 2), then set chunk size to the number of leftover elements, otherwise
    // set chunk_size to the usual (full) number of elements (block_size * 2)

    // Declare an array in shared memory. All threads in a block will have access to this
    // array. The size will be (chunk_size / 2), which is a maximum of 1024 / 2 = 512.
    // The reason we don't need the full chunk_size space is because we'll do an extra step
    // when we transfer our data from global to shared memory: first, we'll read half of it
    // (chunk_size / 2 elements) from global memory and store it in the shared array. Then, we'll
    // read the other half and add it into the existing values in the shared array.
    // In other words, we'll do the first step of our usual for loop in advance. 
    // This means our for loop can be run for one fewer iteration than usual.
    __shared__ int shared[512]; // Note: ideally, the size here would read chunk_size / 2,
                                  // but CUDA forces us to use a constant (512) so the compiler
                                  // can deduce how much shared memory will be required at compile time.

    // Calculate the index that this block's chunk of values starts at.
    // As last time, each thread adds 2 values, so each block adds a total of
    // block_size * 2 values.
    unsigned int block_start = block_id * block_size * 2 + thread_id;

    // Copy half the data from our chunk into shared memory, then add in the other half
    // (as described above).
    if (thread_id < chunk_size / 2)
    {
        shared[thread_id] = input[block_start] + input[block_start + chunk_size / 2];
    }
    // Since shared memory is shared by all warps running on a block (which may
    // not be synchronized), we need to sync here to make sure everybody finishes
    // the above copy before we move on.
    __syncthreads();

    // Perform the rest of the reduction, using the shared memory array.
    // Note that the starting stride is divided by 4 instead of by 2 like we've done in the past.
    // This reflects the fact that we already did one step of the
    // reduction when we copied the data to shared memory above.
    for (unsigned int stride = chunk_size / 4; stride > 0; stride /= 2)
    {
        // we may be running more threads than we need
        if (thread_id < stride)
        {
            shared[thread_id] += shared[thread_id + stride];
        }
        // still need to sync here as usual
        __syncthreads();
    }

    // Thread 0 writes this block's partial result to the output buffer.
    // This time that means we need to copy from *shared memory*
    // back to global memory.
    // The partial result will be in shared array index 0 (remember, 
    // there is a *separate* shared array allocated for each block).
    if (!thread_id)
    {
        output[block_id] = shared[0];
    }
}
*/

int main(int argc, char *argv[]) {
  int fileStarts;
  bool get_c; bool get_w; bool get_l;
  if (argv[1][0] != '-') {
    get_c = true;
    get_w = true;
    get_l = true;
    fileStarts = 1;
  } else {
    get_c = false;
    get_w = false;
    get_l = false;
    for (int i = 1; i < argc; i++) {
      if (argv[i][0] == '-') {
        int j = 1;
        while (argv[i][j] != '\0') {
          switch (argv[i][j]) {
            case 'c':
              get_c = true;
              break;
            case 'w':
              get_w = true;
              break;
            case 'l':
              get_l = true;
              break;
            default:
              std::cout << "Unrecognized option : -" << argv[i][j] << std::endl;
              return 1;
          }
          j++;
        }
      } else {
        fileStarts = i;
        break;
      }
    }
  }
  std::cout << "c, w, l -> " << get_c << " " << get_w << " " << get_l << std::endl;
  std::cout << "fileStarts -> " << fileStarts << " " << argv[fileStarts] << std::endl;
    
  char string[] = "this \n is \n a \n \n test \ntew";
  //int vals[10] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  size_t mem_size = sizeof(char) * 33;
  char* d_in;
  int* d_out;
  hipMalloc((void**)&d_in, mem_size);
  hipMalloc((void**)&d_out, mem_size);
  hipMemcpy(d_in, string, mem_size, hipMemcpyHostToDevice);
  reduce0<<<1, 512>>>(d_in, d_out);
  hipDeviceSynchronize();
  int* h_out = (int*) malloc(sizeof(int) * 1);
  hipMemcpy(h_out, d_out, mem_size, hipMemcpyDeviceToHost);
  std::cout << h_out[0] << std::endl;
}

